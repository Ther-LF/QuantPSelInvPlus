
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <hipblas.h>
#include <hiprand/hiprand.h>


//Print matrix A(nr_rows_A, nr_cols_A) storage in column-major format
void print_matrix(const float *A, int nr_rows_A, int nr_cols_A) {

    for(int i = 0; i < nr_rows_A; ++i){
        for(int j = 0; j < nr_cols_A; ++j){
            std::cout << A[j * nr_rows_A + i] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}

// Multiply the arrays A and B on GPU and save the result in C
// C(m,n) = A(m,k) * B(k,n)
void gpu_blas_mmul( char transA, char transB, int m, int n, int k, 
  float alpha, const float* A, int lda, const float* B, int ldb,
  float beta,        float* C, int ldc ){

	// Allocate 3 arrays on GPU
	float *d_A, *d_B, *d_C;
	hipMalloc(&d_A,m * k * sizeof(float));
	hipMalloc(&d_B,k * n * sizeof(float));
	hipMalloc(&d_C,m * n * sizeof(float));

    // Copy the data to device
    hipMemcpy(d_A, A, m * k * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, n * k * sizeof(float), hipMemcpyHostToDevice);



	// Create a handle for CUBLAS
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasOperation_t opA = HIPBLAS_OP_N;
	hipblasOperation_t opB = HIPBLAS_OP_N;
	if(transA == 'T'){
		opA = HIPBLAS_OP_T;
	}
	if(transB == 'T'){
		opB = HIPBLAS_OP_T;
	}
	
	// Do the actual multiplication
	hipblasSgemm(handle, opA, opB, m, n, k, &alpha, d_A, lda, d_B, ldb, &beta, d_C, ldc);
	// Destroy the handle
	hipblasDestroy(handle);

	// Copy (and print) the result on host memory
	hipMemcpy(C,d_C,m * n * sizeof(float),hipMemcpyDeviceToHost);

	//Free GPU memory
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);	
}

void gpu_blas_trsm( char side, char uplo, char trans, char unit, int m, int n,
  float alpha, const float* A, int lda, float* B, int ldb ){

	// settings in gpu
	hipblasSideMode_t cuSide;
	hipblasFillMode_t cuUplo;
	hipblasOperation_t cuTrans;
	hipblasDiagType_t cuUnit;

	int rowA, colA, rowB = m, colB = n;
	// modify the settings
	if(side == 'L'){
		rowA = lda;
		colA = m;
		cuSide = HIPBLAS_SIDE_LEFT;
	}else if(side == 'R'){
		rowA = lda;
		colA = n;
		cuSide = HIPBLAS_SIDE_RIGHT;
	}


	if(uplo == 'L'){
		cuUplo = HIPBLAS_FILL_MODE_LOWER;
	}else if(uplo == 'U'){
		cuUplo = HIPBLAS_FILL_MODE_UPPER;
	}else{ //这里不知道FULL是什么字符就用了else了，后面会补上的
		cuUplo = HIPBLAS_FILL_MODE_FULL;
	}

	if(trans == 'T'){
		cuTrans = HIPBLAS_OP_T;
	}else if(trans == 'N'){
		cuTrans = HIPBLAS_OP_N;
	}

	if(unit == 'U'){
		cuUnit = HIPBLAS_DIAG_UNIT;
	}else{ //这里不知道NON_UNIT是什么字符就用了else了，后面会补上的
		cuUnit = HIPBLAS_DIAG_NON_UNIT;
	}

	// Allocate 3 arrays on GPU
	float *d_A, *d_B;
	hipMalloc(&d_A,rowA * colA * sizeof(float));
	hipMalloc(&d_B,rowB * colB * sizeof(float));

	// Copy the data to device
    hipMemcpy(d_A, A, rowA * colA * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, rowB * colB * sizeof(float), hipMemcpyHostToDevice);

	// Create a handle for CUBLAS
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	// Do the actual trsm
	hipblasStrsm(handle, cuSide, cuUplo, cuTrans, cuUnit, m, n, &alpha, d_A, lda, d_B, ldb);
	// Destroy the handle
	hipblasDestroy(handle);

	// Copy (and print) the result on host memory
	hipMemcpy(B, d_B, m * n * sizeof(float), hipMemcpyDeviceToHost);

	//Free GPU memory
	hipFree(d_A);
	hipFree(d_B);
}
