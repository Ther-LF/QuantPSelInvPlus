#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <hipblas.h>
#include <hiprand.h>
#include <cstdlib>
#include <ctime>


extern "C" void strsm_( char side, char uplo, char trans, char unit, int m, int n,
  float alpha, const float* A, int lda, float* B, int ldb );
// static hipblasHandle_t handle;
// void init_handle(){
// 	hipblasCreate(&handle);
// }
// void del_handle(){
// 	hipblasDestroy(handle);
// }
//Print matrix A(nr_rows_A, nr_cols_A) storage in column-major format
const int CLOCK_RATE = 1410000;
void print_matrix(const float *A, int nr_rows_A, int nr_cols_A) {

    for(int i = 0; i < nr_rows_A; ++i){
        for(int j = 0; j < nr_cols_A; ++j){
            std::cout << A[j * nr_rows_A + i] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}
// Multiply the arrays A and B on GPU and save the result in C
// C(m,n) = A(m,k) * B(k,n)
void gpu_blas_mmul(const float *A, const float *B, float *C, const int m, const int k, const int n) {
	int lda=m,ldb=k,ldc=m;
    const float alf = 1;
    const float bet = 0;
    const float *alpha = &alf;
    const float *beta = &bet;
	// Allocate 3 arrays on GPU
	float *d_A, *d_B, *d_C;
	hipMalloc(&d_A,m * k * sizeof(float));
	hipMalloc(&d_B,k * n * sizeof(float));
	hipMalloc(&d_C,m * n * sizeof(float));

    // Copy the data to device
    hipMemcpy(d_A, A, m * k * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, n * k * sizeof(float), hipMemcpyHostToDevice);

    // Print the matrix
    std::cout << "A =" << std::endl;
	print_matrix(A, m, k);
	std::cout << "B =" << std::endl;
	print_matrix(B, k, n);



	// Create a handle for CUBLAS
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	// hipblasOperation_t opA = HIPBLAS_OP_N;
	// hipblasOperation_t opB = HIPBLAS_OP_N;
	// if(transA == 'T'){
	// 	opA = HIPBLAS_OP_T;
	// }
	// if(transB == 'T'){
	// 	opB = HIPBLAS_OP_T;
	// }
	
	// Do the actual multiplication
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, d_A, lda, d_B, ldb, beta, d_C, ldc);
	// Destroy the handle
	hipblasDestroy(handle);

	// Copy (and print) the result on host memory
	hipMemcpy(C,d_C,m * n * sizeof(float),hipMemcpyDeviceToHost);
	std::cout << "C =" << std::endl;
	print_matrix(C, m, n);

	//Free GPU memory
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);	
}

void gpu_blas_trsm( char side, char uplo, char trans, char unit, int m, int n,
  float alpha, const float* A, int lda, float* B, int ldb ){

	// settings in gpu
	hipblasSideMode_t cuSide;
	hipblasFillMode_t cuUplo;
	hipblasOperation_t cuTrans;
	hipblasDiagType_t cuUnit;

	int rowA, colA, rowB = m, colB = n;
	// modify the settings
	if(side == 'L'){
		rowA = lda;
		colA = m;
		cuSide = HIPBLAS_SIDE_LEFT;
	}else if(side == 'R'){
		rowA = lda;
		colA = n;
		cuSide = HIPBLAS_SIDE_RIGHT;
	}


	if(uplo == 'L'){
		cuUplo = HIPBLAS_FILL_MODE_LOWER;
	}else if(uplo == 'U'){
		cuUplo = HIPBLAS_FILL_MODE_UPPER;
	}else{ //这里不知道FULL是什么字符就用了else了，后面会补上的
		cuUplo = HIPBLAS_FILL_MODE_FULL;
	}

	if(trans == 'T'){
		cuTrans = HIPBLAS_OP_T;
	}else if(trans == 'N'){
		cuTrans = HIPBLAS_OP_N;
	}

	if(unit == 'U'){
		cuUnit = HIPBLAS_DIAG_UNIT;
	}else{ //这里不知道NON_UNIT是什么字符就用了else了，后面会补上的
		cuUnit = HIPBLAS_DIAG_NON_UNIT;
	}

	// Allocate 3 arrays on GPU
	float *d_A, *d_B;
	hipMalloc(&d_A,rowA * colA * sizeof(float));
	hipMalloc(&d_B,rowB * colB * sizeof(float));

	// Copy the data to device
    hipMemcpy(d_A, A, rowA * colA * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, rowB * colB * sizeof(float), hipMemcpyHostToDevice);

	// Create a handle for CUBLAS
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	// Do the actual trsm
	hipblasStrsm(handle, cuSide, cuUplo, cuTrans, cuUnit, m, n, &alpha, d_A, lda, d_B, ldb);
	// Destroy the handle
	hipblasDestroy(handle);

	// Copy (and print) the result on host memory
	hipMemcpy(B, d_B, m * n * sizeof(float), hipMemcpyDeviceToHost);

	//Free GPU memory
	hipFree(d_A);
	hipFree(d_B);
}


void printDeviceProp(const hipDeviceProp_t &prop)
{
    printf("Device Name : %s.\n", prop.name);
    printf("totalGlobalMem : %lu.\n", prop.totalGlobalMem);
    printf("sharedMemPerBlock : %lu.\n", prop.sharedMemPerBlock);
    printf("regsPerBlock : %d.\n", prop.regsPerBlock);
    printf("warpSize : %d.\n", prop.warpSize);
    printf("memPitch : %lu.\n", prop.memPitch);
    printf("maxThreadsPerBlock : %d.\n", prop.maxThreadsPerBlock);
    printf("maxThreadsDim[0 - 2] : %d %d %d.\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf("maxGridSize[0 - 2] : %d %d %d.\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    printf("totalConstMem : %lu.\n", prop.totalConstMem);
    printf("major.minor : %d.%d.\n", prop.major, prop.minor);
    printf("clockRate : %d.\n", prop.clockRate);
    printf("textufloatignment : %lu.\n", prop.textureAlignment);
    printf("deviceOverlap : %d.\n", prop.deviceOverlap);
    printf("multiProcessorCount : %d.\n", prop.multiProcessorCount);
}

int main(){
	int rowB = 15000, colB = 10000, rowA = 10000, colA = 10000;
	float* A = (float*)malloc(sizeof(float) * rowA * colA);
	float* B = (float*)malloc(sizeof(float) * rowB * colB);
	memset(A, 0, sizeof(float) * rowA * colA);
	for(int i = 0;i < rowA;i++){
		for(int j = 0;j<i;j++){
			A[i + j * rowA] = (float)(rand() / double(RAND_MAX));
		}
		A[i + i * rowA] = 1;
	}
	for(int i = 0;i < rowB; i++){
		for(int j = 0; j < colB; j++){
			B[i + j * rowB] = (float)(rand() / double(RAND_MAX));
		}
	}
	// print_matrix(A, rowA, colA);
	// print_matrix(B, rowB, colB);
	int start = clock();
	gpu_blas_trsm('R', 'L', 'N', 'U', rowB, colB, 1.0, A, colB, B, rowB);
	int cost = clock() - start;
	// print_matrix(B, rowB, colB);
	printf("\ntime cost : %f\n", cost * 1.0 / CLOCK_RATE);

	free(A);
	free(B);
	return 0;
}